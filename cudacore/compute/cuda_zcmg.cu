#include "hip/hip_runtime.h"
/**
 *
 * Copyright (c) 2017-2021  King Abdullah University of Science and Technology
 * All rights reserved.
 *
 * ExaGeoStat is a software package provided by KAUST
 **/
/**
 *
 * @file cuda_zcmg.c
 *
 * CUDA Covariance Matrix Generation
 *
 * @version 1.2.0
 *
 * @author Sameh Abdulah
 * @date 2020-06-06
 *
 **/

#define CHUNKSIZE 32

#include <hipblas.h>
#include <stdio.h>
#include "../include/exageostatcudacore.h"

__global__ void dcmg_powexp_kernel(double *A, int m, int n, int m0,
        int n0, double* l1_x_cuda, double* l1_y_cuda, double* l2_x_cuda, double* l2_y_cuda,
        double localtheta0, double localtheta1, double localtheta2, int distance_metric)
    //! Power-Exponential Kernel
    /*!
     * Returns covariance matrix tile using the aforementioned kernel.
     * @param[in] A: 1D array which saves the matrix entries by column.
     * @param[in] m: number of rows of tile.
     * @param[in] n: number of columns of tile.
     * @param[in] m0: Global row start point of tile.
     * @param[in] n0: Global column start point of tile.
     * @param[in] l1_x_cuda: value of x-axis of locaton vector l1.
     * @param[in] l1_y_cuda: value of y-axis of locaton vector l1.
     * @param[in] l2_x_cuda: value of x-axis of locaton vector l2.
     * @param[in] l2_y_cuda: value of y-axis of locaton vector l2.
     * @param[in] localtheta: there are three parameters to define this kernel.
     * @param[in] distance_metric: unused arguments
     * */
{
   int idx = blockIdx.x * blockDim.x + threadIdx.x;
   int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if(idx >= m || idy >= n) return;

    double expr  = 0.0;
    double expr1 = 0.0;
    double sigma_square = localtheta0;
    expr = sqrt(pow((l2_x_cuda[idy] - l1_x_cuda[idx]), 2) +
            pow((l2_y_cuda[idy] - l1_y_cuda[idx]), 2));
    expr1 = pow(expr, localtheta2);
    if(expr == 0){
        A[idx + idy * m] = sigma_square /*+ 1e-4*/;
    }
    else{
        A[idx + idy * m] = sigma_square *  exp(-(expr1/localtheta1)); // power-exp kernel
    }
    

}

void dcmg_powexp( double *A, int m, int n, int m0,
        int n0, double* l1_x_cuda, double* l1_y_cuda, double* l2_x_cuda, double* l2_y_cuda,
        double *localtheta, int distance_metric, hipStream_t stream){

    int nBlockx= (m+CHUNKSIZE-1)/CHUNKSIZE;
    int nBlocky= (n+CHUNKSIZE-1)/CHUNKSIZE;
    dim3 dimBlock(CHUNKSIZE,CHUNKSIZE);
    dim3 dimGrid(nBlockx,nBlocky);

    dcmg_powexp_kernel<<<dimGrid, dimBlock, 0, stream>>>(A, m, n, m0, n0, l1_x_cuda, l1_y_cuda, l2_x_cuda, l2_y_cuda, localtheta[0],localtheta[1],localtheta[2], distance_metric);

    hipStreamSynchronize(stream);

}


// TO-DO: Matern kernel implementation
// __global__ void dcmg_matern_kernel(double *A, int m, int n, int m0,
//         int n0, double* l1_x_cuda, double* l1_y_cuda, double* l2_x_cuda, double* l2_y_cuda,
//         double localtheta0, double localtheta1, double localtheta2, int distance_metric)
//     //! Matern Kernel
//     /*!
//      * Returns covariance matrix tile using the aforementioned kernel.
//      * @param[in] A: 1D array which saves the matrix entries by column.
//      * @param[in] m: number of rows of tile.
//      * @param[in] n: number of columns of tile.
//      * @param[in] m0: Global row start point of tile.
//      * @param[in] n0: Global column start point of tile.
//      * @param[in] l1_x_cuda: value of x-axis of locaton vector l1.
//      * @param[in] l1_y_cuda: value of y-axis of locaton vector l1.
//      * @param[in] l2_x_cuda: value of x-axis of locaton vector l2.
//      * @param[in] l2_y_cuda: value of y-axis of locaton vector l2.
//      * @param[in] localtheta: there are three parameters to define this kernel.
//      * @param[in] distance_metric: unused arguments
//      * */
// {
//    int idx = blockIdx.x * blockDim.x + threadIdx.x;
//    int idy = blockIdx.y * blockDim.y + threadIdx.y;

//     if(idx >= m || idy >= n) return;

//     double expr  = 0.0;
//     double expr1 = 0.0;
//     double sigma_square = localtheta0;
//     expr = sqrt(pow((l2_x_cuda[idy] - l1_x_cuda[idx]), 2) +
//             pow((l2_y_cuda[idy] - l1_y_cuda[idx]), 2));

//     expr1 = pow(expr, localtheta2);
//     if(expr == 0){
//         A[idx + idy * m] = sigma_square /*+ 1e-4*/;
//     }
//     else{
//         A[idx + idy * m] = sigma_square *  exp(-(expr1/localtheta1)); // power-exp kernel
//     }
    

// }

// void dcmg_matern( double *A, int m, int n, int m0,
//         int n0, double* l1_x_cuda, double* l1_y_cuda, double* l2_x_cuda, double* l2_y_cuda,
//         double *localtheta, int distance_metric, hipStream_t stream){

//     int nBlockx= (m+CHUNKSIZE-1)/CHUNKSIZE;
//     int nBlocky= (n+CHUNKSIZE-1)/CHUNKSIZE;
//     dim3 dimBlock(CHUNKSIZE,CHUNKSIZE);
//     dim3 dimGrid(nBlockx,nBlocky);

//     dcmg_matern_kernel<<<dimGrid, dimBlock, 0, stream>>>(A, m, n, m0, n0, l1_x_cuda, l1_y_cuda, l2_x_cuda, l2_y_cuda, localtheta[0],localtheta[1],localtheta[2], distance_metric);

//     hipStreamSynchronize(stream);

// }
